
#include <hip/hip_runtime.h>
/*! \file cooling_cuda.cu
 *  \brief Functions to calculate cooling rate for a given rho, P, dt. */

#ifdef CUDA
#ifdef COOLING_GPU

#include<cuda.h>
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"io.h"
#include"cooling_cuda.h"



/*! \fn void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma)
 *  \brief When passed an array of conserved variables and a timestep, adjust the value
           of the total energy for each cell according to the specified cooling function. */
//__global__ void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma, cudaTextureObject_t coolTexObj, cudaTextureObject_t heatTexObj)
__global__ void cooling_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma, Real *dt_array)
{
  __shared__ Real min_dt[TPB];

  int n_cells = nx*ny*nz;
  
  Real d, E;
  Real n, T, T_init;
  Real del_T, dt_sub;
  Real mu; // mean molecular weight
  Real cool; //cooling rate per volume, erg/s/cm^3
  #ifndef DE
  Real vx, vy, vz, p;
  #endif
  #ifdef DE
  Real ge;
  #endif
  //Real T_min = 1.0e4; // minimum temperature allowed
  Real T_min = 0.0; // minimum temperature allowed

  mu = 0.6;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int id = threadIdx.x + blockId * blockDim.x;
  int zid = id / (nx*ny);
  int yid = (id - zid*nx*ny) / nx;
  int xid = id - zid*nx*ny - yid*nx;
  // and a thread id withing hte block
  int tid = threadIdx.x;

  // set min dt to current hydro timestep
  min_dt[tid] = dt;
  __syncthreads();
  

  // all threads do the calculation
  if (xid < nx && yid < ny && zid < nz) {

    // load values of density and pressure
    d  =  dev_conserved[            id];
    E  =  dev_conserved[4*n_cells + id];
    #ifndef DE
    vx =  dev_conserved[1*n_cells + id] / d;
    vy =  dev_conserved[2*n_cells + id] / d;
    vz =  dev_conserved[3*n_cells + id] / d;
    p  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    p  = fmax(p, (Real) TINY_NUMBER);
    #endif
    #ifdef DE
    ge = dev_conserved[5*n_cells + id] / d;
    ge = fmax(ge, (Real) TINY_NUMBER);
    #endif
    
    // calculate the number density of the gas (in cgs)
    n = d*DENSITY_UNIT / (mu * MP);

    // calculate the temperature of the gas
    #ifndef DE
    T_init = p*PRESSURE_UNIT/ (n*KB);
    #endif
    #ifdef DE
    T_init = ge*(gamma-1.0)*SP_ENERGY_UNIT*mu*MP/KB;
    #endif

    // calculate cooling rate per volume
    T = T_init;

    // call the cooling function (could choose primoridial cool)
    //cool = Schure_cool(n, T); 
    cool = Wiersma_cool(n, T); 
    //cool = primordial_cool(n, T);
    //cool = Cloudy_cool(n, T, coolTexObj, heatTexObj);
    
    // calculate change in temperature given dt
    del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);

    // limit change in temperature to 1%
    while (del_T/T > 0.01) {
      // what dt gives del_T = 0.01*T?
      dt_sub = 0.01*T*n*KB/(cool*TIME_UNIT*(gamma-1.0));
      // apply that dt
      T -= cool*dt_sub*TIME_UNIT*(gamma-1.0)/(n*KB);
      // how much time is left from the original timestep?
      dt -= dt_sub;
      // calculate cooling again
      //cool = Schure_cool(n, T);
      cool = Wiersma_cool(n, T);
      //cool = primordial_cool(n, T);
      //cool = Cloudy_cool(n, T, coolTexObj, heatTexObj);
      // calculate new change in temperature
      del_T = cool*dt*TIME_UNIT*(gamma-1.0)/(n*KB);
    }

    // calculate final temperature
    T -= del_T;
    //if (T < 1000) printf("%3d %3d %3d Low T cell. T_init: %e T: %e\n", xid, yid, zid, T_init, T);

    // set a temperature floor
    if (T < T_min) { 
      T = T_min;
    }

    // adjust value of energy based on total change in temperature
    del_T = T_init - T; // total change in T
    E -= n*KB*del_T / ((gamma-1.0)*ENERGY_UNIT);
    #ifdef DE
    ge -= KB*del_T / (mu*MP*(gamma-1.0)*SP_ENERGY_UNIT);
    #endif
    if (del_T/T_init > 0.1) {
      printf("%3d %3d %3d Cooling over 10 percent in hydro dt. T_init: %e T: %e\n", xid, yid, zid, T_init, T);
    }
    // calculate cooling rate for new T
    cool = Wiersma_cool(n, T);
    // limit the timestep such that delta_T is 10% 
    min_dt[tid] = 0.1*T*n*KB/(cool*TIME_UNIT*(gamma-1.0));

    // and send back from kernel
    dev_conserved[4*n_cells + id] = E;
    #ifdef DE
    dev_conserved[5*n_cells + id] = d*ge;
    #endif

  }
  __syncthreads();

  // do the reduction in shared memory (find the min timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      min_dt[tid] = fmin(min_dt[tid], min_dt[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dt_array[blockIdx.x] = min_dt[0];
  

}


/* \fn __device__ Real test_cool(Real n, Real T)
 * \brief Cooling function from Creasey 2011. */
__device__ Real test_cool(int tid, Real n, Real T)
{
  Real T0, T1, lambda, cool;
  T0 = 10000.0;
  T1 = 20*T0;
  cool = 0.0;
  //lambda = 5.0e-24; //cooling coefficient, 5e-24 erg cm^3 s^-1
  lambda = 5.0e-20; //cooling coefficient, 5e-24 erg cm^3 s^-1

  // constant cooling rate 
  //cool = n*n*lambda;

  // Creasey cooling function
  if (T >= T0 && T <= 0.5*(T1+T0)) {
    cool = n*n*lambda*(T - T0) / T0;
  }
  if (T >= 0.5*(T1+T0) && T <= T1) {
    cool = n*n*lambda*(T1 - T) / T0;
  }
 

  //printf("%d %f %f\n", tid, T, cool);
  return cool;

}


/* \fn __device__ Real primordial_cool(Real n, Real T)
 * \brief Primordial hydrogen/helium cooling curve 
          derived according to Katz et al. 1996. */
__device__ Real primordial_cool(Real n, Real T)
{
  Real n_h, Y, y, g_ff, cool;
  Real n_h0, n_hp, n_he0, n_hep, n_hepp, n_e, n_e_old; 
  Real alpha_hp, alpha_hep, alpha_d, alpha_hepp, gamma_eh0, gamma_ehe0, gamma_ehep;
  Real le_h0, le_hep, li_h0, li_he0, li_hep, lr_hp, lr_hep, lr_hepp, ld_hep, l_ff;
  Real gamma_lh0, gamma_lhe0, gamma_lhep, e_h0, e_he0, e_hep, H;
  int heat_flag, n_iter;
  Real diff, tol;

  // set flag to 1 for photoionization & heating
  heat_flag = 0;

  //Real X = 0.76; //hydrogen abundance by mass
  Y = 0.24; //helium abundance by mass
  y = Y/(4 - 4*Y);  

  // set the hydrogen number density 
  n_h = n; 

  // calculate the recombination and collisional ionziation rates
  // (Table 2 from Katz 1996)
  alpha_hp   = (8.4e-11) * (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  alpha_hep  = (1.5e-10) * (pow(T,(-0.6353)));
  alpha_d    = (1.9e-3)  * (pow(T,(-1.5))) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T));
  alpha_hepp = (3.36e-10)* (1.0/sqrt(T)) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7))));
  gamma_eh0  = (5.85e-11)* sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehe0 = (2.38e-11)* sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  gamma_ehep = (5.68e-12)* sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5)));
  // externally evaluated integrals for photoionziation rates
  // assumed J(nu) = 10^-22 (nu_L/nu)
  gamma_lh0 = 3.19851e-13;
  gamma_lhe0 = 3.13029e-13;
  gamma_lhep = 2.00541e-14; 
  // externally evaluated integrals for heating rates
  e_h0 = 2.4796e-24;
  e_he0 = 6.86167e-24;
  e_hep = 6.21868e-25; 
  

  // assuming no photoionization, solve equations for number density of
  // each species
  n_e = n_h; //as a first guess, use the hydrogen number density
  n_iter = 20;
  diff = 1.0;
  tol = 1.0e-6;
  if (heat_flag) { 
    for (int i=0; i<n_iter; i++) {
      n_e_old = n_e;
      n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0 + gamma_lh0/n_e);
      n_hp   = n_h - n_h0;
      n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0 + gamma_lhe0/n_e) + (gamma_ehep + gamma_lhep/n_e)/alpha_hepp );
      n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0 + gamma_lhe0/n_e);
      n_hepp = n_hep*(gamma_ehep + gamma_lhep/n_e)/alpha_hepp;
      n_e    = n_hp + n_hep + 2*n_hepp;
      diff = fabs(n_e_old - n_e);
      if (diff < tol) break;
    }
  }  
  else {
    n_h0   = n_h*alpha_hp / (alpha_hp + gamma_eh0);
    n_hp   = n_h - n_h0;
    n_hep  = y*n_h / (1.0 + (alpha_hep + alpha_d)/(gamma_ehe0) + (gamma_ehep)/alpha_hepp );
    n_he0  = n_hep*(alpha_hep + alpha_d) / (gamma_ehe0);
    n_hepp = n_hep*(gamma_ehep)/alpha_hepp;
    n_e    = n_hp + n_hep + 2*n_hepp;
  }

  // using number densities, calculate cooling rates for
  // various processes (Table 1 from Katz 1996)
  le_h0 = (7.50e-19) * exp(-118348.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  le_hep = (5.54e-17) * pow(T,(-0.397)) * exp(-473638.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  li_h0 = (1.27e-21) * sqrt(T) * exp(-157809.1/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_h0;
  li_he0 = (9.38e-22) * sqrt(T) * exp(-285335.4/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_he0;
  li_hep = (4.95e-22) * sqrt(T) * exp(-631515.0/T) * (1.0 / (1.0 + sqrt(T/1e5))) * n_e * n_hep;
  lr_hp = (8.70e-27) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hp;
  lr_hep = (1.55e-26) * pow(T,(0.3647)) * n_e * n_hep;
  lr_hepp = (3.48e-26) * sqrt(T) * pow((T/1e3),(-0.2)) * (1.0 / (1.0 + pow((T/1e6),(0.7)))) * n_e * n_hepp;
  ld_hep = (1.24e-13) * pow(T,(-1.5)) * exp(-470000.0/T) * (1.0 + 0.3*exp(-94000.0/T)) * n_e * n_hep;
  g_ff = 1.1 + 0.34*exp(-(5.5-log(T))*(5.5-log(T))/3.0); // Gaunt factor
  l_ff = (1.42e-27) * g_ff * sqrt(T) * (n_hp + n_hep + 4*n_hepp) * n_e;

  // calculate total cooling rate (erg s^-1 cm^-3)
  cool = le_h0 + le_hep + li_h0 + li_he0 + li_hep + lr_hp + lr_hep + lr_hepp + ld_hep + l_ff;

  // calculate total photoionization heating rate
  H = 0.0;
  if (heat_flag) {
    H = n_h0*e_h0 + n_he0*e_he0 + n_hep*e_hep; 
  }
  
  cool -= H;

  return cool;

}



/* \fn __device__ Real Schure_cool(Real n, Real T)
 * \brief Analytic fit to the solar metallicity CIE cooling curve 
          defined in Schure et al., 2009. */
__device__ Real Schure_cool(Real n, Real T)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  
  // fit to Schure cooling function 
  if (log10(T) > 5.36) {
    lambda = pow(10.0, (0.38 * (log10(T) -7.5) * (log10(T) - 7.5) - 22.6));
  }
  else if (log10(T) < 4.0) {
    lambda = 0.0;
  }
  else {
    lambda = pow(10.0, (-2.5 * (log10(T) - 5.1) * (log10(T) - 5.1) - 20.7));
  }

  // cooling rate per unit volume
  cool = n*n*lambda;

  return cool;

}

/* \fn __device__ Real Wiersma_cool(Real n, Real T)
 * \brief Analytic fit to the solar metallicity CIE cooling curve 
          defined in Wiersma et al., 2009. */
__device__ Real Wiersma_cool(Real n, Real T)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  
  // fit to Wiersma 2009 CIE cooling function 
  if (log10(T) < 4.0) {
    lambda = 0.0;
  }
  else if (log10(T) >= 4.0 && log10(T) < 5.9) {
    lambda = pow(10.0, (-1.3 * (log10(T) - 5.25) * (log10(T) - 5.25) - 21.25));
  }
  else if (log10(T) >= 5.9 && log10(T) < 7.4) {
    lambda = pow(10.0, (0.7 * (log10(T) - 7.1) * (log10(T) - 7.1) - 22.8));
  }
  else {
    lambda = pow(10.0, (0.45*log10(T) - 26.065));
  }

  // cooling rate per unit volume
  cool = n*n*lambda;

  return cool;

}


/* \fn __device__ Real Cloudy_cool(Real n, Real T, cudaTextureObject_t coolTexObj, cudaTextureObject_t heatTexObj)
 * \brief Uses texture mapping to interpolate Cloudy cooling/heating 
          tables at z = 0 with solar metallicity and an HM05 UV background. */
__device__ Real Cloudy_cool(Real n, Real T, cudaTextureObject_t coolTexObj, cudaTextureObject_t heatTexObj)
{
  Real lambda = 0.0; //cooling rate, erg s^-1 cm^3
  Real H = 0.0; //heating rate, erg s^-1 cm^3
  Real cool = 0.0; //cooling per unit volume, erg /s / cm^3
  float log_n, log_T;
  log_n = log10(n);
  log_T = log10(T);

  // don't allow cooling at super low temps
  if (log_T < 1.0) return cool;

/*
  // use analytic curve for cooling 
  if (log10(T) < 4.0) {
    lambda = 0.0;
  }
  else if (log10(T) >= 4.0 && log10(T) < 5.9) {
    lambda = pow(10.0, (-1.3 * (log10(T) - 5.25) * (log10(T) - 5.25) - 21.25));
  }
  else if (log10(T) >= 5.9 && log10(T) < 7.4) {
    lambda = pow(10.0, (0.7 * (log10(T) - 7.1) * (log10(T) - 7.1) - 22.8));
  }
  else {
    lambda = pow(10.0, (0.45*log10(T) - 26.065));
  }
*/

  // keep estimates within the bounds of the textures
  // this is done automatically by setting cudaAddressModeClamp
  //log_T = fmin(log_T, 9.0);
  //log_n = fmax(log_n, -6.0);
  //log_n = fmin(log_n, 6.0);

  // remap coordinates for texture
  log_T = (log_T - 1.0)/8.1;
  log_n = (log_n + 6.0)/12.1; 
 
  if (log10(T) > 4.0) {
  lambda = tex2D<float>(coolTexObj, log_T, log_n);
  }
  else lambda = 0.0;
  H = tex2D<float>(heatTexObj, log_T, log_n);

  // cooling rate per unit volume
  cool = n*n*(powf(10, lambda) - powf(10, H));

  return cool;
}

#endif //COOLING_GPU
#endif //CUDA
